#include "hip/hip_runtime.h"
extern "C++" {
    #include "stdlib.h"
    #include "stdio.h"
}

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


__device__ unsigned int
    d_input_height, d_input_width,
    d_filter_height, d_filter_width,
    d_output_height, d_output_width;


#define cudaErrorCheck(ans) { cudaAssert((ans), __FILE__, __LINE__); }


inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}


void print_matrix(const char *name, float *matrix, int height, int width) {
    printf("%s\n", name);
    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            printf(" %5.2f", matrix[width * y + x]);
        }
        printf("\n");
    }
    printf("\n");
}


__global__
static int POSITION(x, y, width) {
    return width * y + x;
}


__global__
void gpu__convolute(float *input, float *filter, float *output, int output_size) {
    // idx = (width) * (y) + (x)
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int width = gridDim.x * blockDim.x
    int idx = POSITION(x, y, width);

    if (idx < output_size) {
        for (int cy = 0; cy < d_filter_height; cy++) {
            for (int cx = 0; cx < d_filter_width; cx++) {
                output[idx] = input[POSITION(x+cx, y+cy, d_input_width)] * filter[POSITION(cx, cy, d_filter_width)];
            }
        }
    }
}


// static int hPOSITION(x, y, width) {
//     return width * y + x;
// }


void usage(char *executable) {
    printf("ERROR, incorrect arguments.\n");
    printf("usage:\n");
    printf("\t %s <input height: int> <input width: int> <filter height: int> <filter width: int>\n", executable);
    exit(1);
}


int main(int argc, char **argv) {
    if (argc != 4) {
        usage(argv[0]);
    }

    // Hard-code for testing output with linear version
    srand48(20171116);

    int input_height = atoi(argv[1]);
    int input_width = atoi(argv[2]);

    int filter_height = atoi(argv[3]);
    int filter_width = atoi(argv[4]);

    int output_height = input_height - filter_height + 1;
    int output_width = input_width - filter_width + 1;

    // Setup input array
    float *cpu__input = (float*) malloc(input_height * input_width * sizeof(float));
    for (int y = 0; y < input_height; y++) {
        for (int x = 0; x < input_width; x++) {
            cpu__input[POSITION(x, y, input_width)] = drand48() * 100;
        }
    }

    // Setup filter array
    float *cpu__filter = (float*) malloc(filter_height * filter_width * sizeof(float));
    for (int y = 0; y < filter_height; y++) {
        for (int x = 0; x < filter_width; x++) {
            cpu__filter[POSITION(x, y, filter_width)] = drand48() * 100;
        }
    }

    // Setup output array
    float *cpu__output = (float*) malloc(output_height * output_width * sizeof(float));
    for (int y = 0; y < output_height; y++) {
        for (int x = 0; x < output_width; x++) {
            cpu__output[POSITION(x, y, output_width)] = 0.0;
        }
    }

    print_matrix("input", input, input_height, input_width);
    print_matrix("filter", filter, filter_height, filter_width);


    hipSetDevice(0);

    // Copy scalar variables onto GPU
    cudaErrorCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_input_height), (void*) &input_height, sizeof(unsigned int)) );
    cudaErrorCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_input_width),  (void*) &input_width,  sizeof(unsigned int)) );

    cudaErrorCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_filter_height), (void*) &filter_height, sizeof(unsigned int)) );
    cudaErrorCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_filter_width),  (void*) &filter_width,  sizeof(unsigned int)) );

    cudaErrorCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_output_height), (void*) &output_height, sizeof(unsigned int)) );
    cudaErrorCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_output_width),  (void*) &output_width,  sizeof(unsigned int)) );

    float
        *gpu__input,
        *gpu__filter,
        *gpu__output;


    // Allocate memory for arrays on GPU
    cudaErrorCheck( hipMalloc((void**) &gpu__input,  input_height  * input_width  * sizeof(float)) );
    cudaErrorCheck( hipMalloc((void**) &gpu__filter, filter_height * filter_width * sizeof(float)) );
    cudaErrorCheck( hipMalloc((void**) &gpu__output, output_height * output_width * sizeof(float)) );

    // Copy memory for arrays from CPU -> GPU
    cudaErrorCheck( hipMemcpy(gpu__input,  cpu__input,  input_height  * input_width  * sizeof(float), hipMemcpyHostToDevice) );
    cudaErrorCheck( hipMemcpy(gpu__filter, cpu__filter, filter_height * filter_width * sizeof(float), hipMemcpyHostToDevice) );
    cudaErrorCheck( hipMemcpy(gpu__output, cpu__output, output_height * output_width * sizeof(float), hipMemcpyHostToDevice) );


    int block_height = 32.0;
    int block_width = 32.0;

    dim3 dimGrid(ceil(output_width / block_width), ceil(output_height / block_height), 1);
    dim3 dimBlock(block_width, block_height, 1);

    convolute<<<dimGride, dimBlock>>>(gpu__input, gpu__filter, gpu__output, d_output_height * d_output_width);

    // Copy memory for arrays from GPU -> CPU
    cudaErrorCheck( hipMemcpy(cpu__output, gpu__output, output_height * output_width * sizeof(float), hipMemcpyDeviceToHost) );


    print_matrix("output", output, output_height, output_width);

    free(cpu__input);  cpu__input  = nullptr;
    free(cpu__filter); cpu__filter = nullptr;
    free(cpu__output); cpu__output = nullptr;

    hipFree(gpu__input);  gpu__input  = nullptr;
    hipFree(gpu__filter); gpu__filter = nullptr;
    hipFree(gpu__output); gpu__output = nullptr;
}
